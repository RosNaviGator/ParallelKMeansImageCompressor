#include "hip/hip_runtime.h"
#include <kMeansCUDA.cuh>

km::KMeansCUDA::KMeansCUDA(const int &k, const std::vector<Point> &points) : k(k), points(points)
{
    int size = points.size();
    std::random_device rd;                            // Initialize a random device
    std::mt19937 gen(rd());                       // Initialize a Mersenne Twister random number generator with the random device
    std::uniform_int_distribution<> dis(0, size - 1); // Create a uniform distribution between 0 and size - 1

    for (int i = 0; i < k; ++i)
    {
        centroids.emplace_back(points[dis(gen)]); // Generate a random index and use it to select a point
    }
}

__global__ void km::KMeansCUDA::assign_clusters(int *data, int *centroids, int *labels, int n, int k, int dim) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    // let's print the centrids
    // for (int i = 0; i < k; i++) {
    //    printf("Centroid %d: %d %d %d\n", i, centroids[i * dim], centroids[i * dim + 1], centroids[i * dim + 2]);
    // }

    int closest = 0;
    int min_dist = 1 << 30;

    for (int i = 0; i < k; ++i) {
        int dist = 0.0f;
        for (int j = 0; j < dim; ++j) {
            int diff = data[idx * dim + j] - centroids[i * dim + j];
            dist += diff * diff;
        }
        if (dist < min_dist) {
            min_dist = dist;
            closest = i;
        }
    }
    labels[idx] = closest;
    //printf("Point %d assigned to cluster %d\n", idx, closest);
}

__global__ void km::KMeansCUDA::calculate_new_centroids(int *data, int *centroids, int *labels, int *counts, int n, int k, int dim) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    int cluster_id = labels[idx];
    for (int j = 0; j < dim; ++j) {
        atomicAdd(&centroids[cluster_id * dim + j], data[idx * dim + j]);
    }
    atomicAdd(&counts[cluster_id], 1);
}

__global__ void km::KMeansCUDA::average_centroids(int *centroids, int *counts, int k, int dim) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= k) return;

    for (int j = 0; j < dim; ++j) {
        if (counts[idx] > 0) {
            centroids[idx * dim + j] /= counts[idx];
        }
    }
}

void km::KMeansCUDA::run()
{
    int MAX_ITER = 100;
    int n = points.size();
    int dim = 3;
    int* labels = new int[n];
    int* centroids_data =  new int[k*dim];
    int* points_data = new int[n * dim];


    for (int i = 0; i < points.size(); i++) 
    {
        for (int j = 0; j < 3; j++) 
        {  
            points_data[i * 3 + j] = points.at(i).getFeature_int(j);   
        }
    }

    for (int i = 0; i < centroids.size(); i++) 
    {
        for (int j = 0; j < 3; j++) 
        {
            centroids_data[i * 3 + j] = centroids.at(i).getFeature_int(j);
        }
    }


    int *d_data, *d_centroids;
    int *d_labels, *d_counts;
    int* temp = new int[k * dim];
    memccpy(temp, centroids_data, sizeof(int) * k * dim, sizeof(int) * k * dim);
    

    hipMalloc(&d_data, sizeof(int) * n * dim);
    hipMalloc(&d_centroids, sizeof(int) * k * dim);
    hipMalloc(&d_labels, n * sizeof(int));
    hipMalloc(&d_counts, k * sizeof(int));

    hipMemcpy(d_data, points_data, sizeof(int) * n * dim, hipMemcpyHostToDevice);
    hipMemcpy(d_centroids, centroids_data, sizeof(int) * k * dim, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocksN = (n + blockSize - 1) / blockSize;
    int numBlocksK = (k + blockSize - 1) / blockSize;

    bool converged = false;

    for (int iter = 0; iter < MAX_ITER || !converged; ++iter) {
        std::cout << "Iteration " << iter << std::endl;
        assign_clusters<<<numBlocksN, blockSize>>>(d_data, d_centroids, d_labels, n, k, dim);
        hipDeviceSynchronize();
        
        hipMemset(d_centroids, 0, sizeof(int) * k * dim);
        hipMemset(d_counts, 0, k * sizeof(int));
        calculate_new_centroids<<<numBlocksN, blockSize>>>(d_data, d_centroids, d_labels, d_counts, n, k, dim);
        hipDeviceSynchronize();

        average_centroids<<<numBlocksK, blockSize>>>(d_centroids, d_counts, k, dim);
        hipDeviceSynchronize();


        int* returnedCentroids;
        returnedCentroids = new int[k * dim];
        hipMemcpy(returnedCentroids, d_centroids, sizeof(int) * k * dim, hipMemcpyDeviceToHost);

        // check if the centroids have changed
        converged = true;
        for (int i = 0; i < k; i++) {
            for (int j = 0; j < dim; j++) {
                if (temp[i * dim + j] != returnedCentroids[i * dim + j]) {
                    converged = false;
                    temp[i * dim + j] = returnedCentroids[i * dim + j];
                    break;
                }
            }
        }

        if (converged)
        {
            number_of_iterations = iter;
            break;
        } 

    }

    hipMemcpy(labels, d_labels, n * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(centroids_data, d_centroids, sizeof(int) * k * dim, hipMemcpyDeviceToHost);

    for (int i = 0 ; i < points.size() ; i++)
    {
        points.at(i).clusterId = labels[i];
        //std::cout << "Point " << i << " clusterId: " << points.at(i).clusterId << std::endl;
    }

    for (int i = 0; i < centroids.size(); i++) 
    {
        for (int j = 0; j < 3; j++) 
        {
            centroids.at(i).setFeature(j, centroids_data[i * 3 + j]);
        }
    }
    hipFree(d_data);
    hipFree(d_centroids);
    hipFree(d_labels);
    hipFree(d_counts);
}

auto km::KMeansCUDA::getCentroids() -> std::vector<Point>
{
    return centroids;
}

auto km::KMeansCUDA::getPoints() -> std::vector<Point>
{
    return points;
}

auto km::KMeansCUDA::getIterations() -> int
{
    return number_of_iterations;
}